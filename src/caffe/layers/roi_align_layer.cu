#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/roi_align_layer.hpp"
#include "caffe/util/gpu_util.cuh"

using std::max;
using std::min;

namespace caffe {
	
	template <typename Dtype>
	__global__ void RoIAlignForward(
		const int nthreads,
		const Dtype* bottom_data,
		const Dtype spatial_scale,
		const int channels,
		const int height,
		const int width,
		const int pooled_height,
		const int pooled_width,
		const int sampling_ratio,
		const Dtype* bottom_rois,
		Dtype* top_data) {
		CUDA_KERNEL_LOOP(index,nthreads) {
			// (n, c, ph, pw) is an element in the pooled output
			int pw = index % pooled_width;
			int ph = (index / pooled_width) % pooled_height;
			int c = (index / pooled_width / pooled_height) % channels;
			int n = index / pooled_width / pooled_height / channels;

			const Dtype* offset_bottom_rois = bottom_rois + n * 5;
			int roi_batch_ind = offset_bottom_rois[0];

			// Do not using rounding; this implementation detail is critical
			Dtype roi_start_w = offset_bottom_rois[1] * spatial_scale;
			Dtype roi_start_h = offset_bottom_rois[2] * spatial_scale;
			Dtype roi_end_w = offset_bottom_rois[3] * spatial_scale;
			Dtype roi_end_h = offset_bottom_rois[4] * spatial_scale;
			// T roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
			// T roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
			// T roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
			// T roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

			// Force malformed ROIs to be 1x1
			Dtype roi_width = max(roi_end_w - roi_start_w, (Dtype)1.);
			Dtype roi_height = max(roi_end_h - roi_start_h, (Dtype)1.);
			Dtype bin_size_h = static_cast<Dtype>(roi_height) / static_cast<Dtype>(pooled_height);
			Dtype bin_size_w = static_cast<Dtype>(roi_width) / static_cast<Dtype>(pooled_width);

			const Dtype* offset_bottom_data =
				bottom_data + (roi_batch_ind * channels + c) * height * width;

			// We use roi_bin_grid to sample the grid and mimic integral
			int roi_bin_grid_h = (sampling_ratio > 0)
				? sampling_ratio
				: ceil(roi_height / pooled_height); // e.g., = 2
			int roi_bin_grid_w =
				(sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

			// We do average (integral) pooling inside a bin
			const Dtype count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

			Dtype output_val = 0.;
			for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
			{
				const Dtype y = roi_start_h + ph * bin_size_h +
					static_cast<Dtype>(iy + .5f) * bin_size_h /
					static_cast<Dtype>(roi_bin_grid_h); // e.g., 0.5, 1.5
				for (int ix = 0; ix < roi_bin_grid_w; ix++) {
					const Dtype x = roi_start_w + pw * bin_size_w +
						static_cast<Dtype>(ix + .5f) * bin_size_w /
						static_cast<Dtype>(roi_bin_grid_w);

					Dtype val = bilinear_interpolate(
						offset_bottom_data, height, width, y, x, index);
					output_val += val;
				}
			}
			output_val /= count;

			top_data[index] = output_val;
		}
	}


template <typename Dtype>
void ROIAlignLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
	const Dtype* bottom_data = bottom[0]->gpu_data();
	const Dtype* bottom_rois = bottom[1]->gpu_data();
	Dtype* top_data = top[0]->mutable_gpu_data();
	int* argmax_data = max_idx_.mutable_gpu_data();
	int count = top[0]->count();
	RoIAlignForward<Dtype> << <CAFFE_GET_BLOCKS(count),
		CAFFE_CUDA_NUM_THREADS >> >(
		count,
		bottom_data,
		spatial_scale_,
		channels_,
		height_,
		width_,
		pooled_height_,
		pooled_width_,
		sampling_ratio_,
		bottom_rois,
		top_data);

  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void RoIAlignBackwardFeature(
	const int nthreads,
	const Dtype* top_diff,
	const int num_rois,
	const Dtype spatial_scale,
	const int channels,
	const int height,
	const int width,
	const int pooled_height,
	const int pooled_width,
	const int sampling_ratio,
	Dtype* bottom_diff,
	const Dtype* bottom_rois) {
	CUDA_KERNEL_LOOP(index, nthreads) {
		// (n, c, ph, pw) is an element in the pooled output
		int pw = index % pooled_width;
		int ph = (index / pooled_width) % pooled_height;
		int c = (index / pooled_width / pooled_height) % channels;
		int n = index / pooled_width / pooled_height / channels;

		const Dtype* offset_bottom_rois = bottom_rois + n * 5;
		int roi_batch_ind = offset_bottom_rois[0];

		// Do not using rounding; this implementation detail is critical
		Dtype roi_start_w = offset_bottom_rois[1] * spatial_scale;
		Dtype roi_start_h = offset_bottom_rois[2] * spatial_scale;
		Dtype roi_end_w = offset_bottom_rois[3] * spatial_scale;
		Dtype roi_end_h = offset_bottom_rois[4] * spatial_scale;
		// T roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
		// T roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
		// T roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
		// T roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

		// Force malformed ROIs to be 1x1
		Dtype roi_width = max(roi_end_w - roi_start_w, (Dtype)1.);
		Dtype roi_height = max(roi_end_h - roi_start_h, (Dtype)1.);
		Dtype bin_size_h = static_cast<Dtype>(roi_height) / static_cast<Dtype>(pooled_height);
		Dtype bin_size_w = static_cast<Dtype>(roi_width) / static_cast<Dtype>(pooled_width);

		Dtype* offset_bottom_diff =
			bottom_diff + (roi_batch_ind * channels + c) * height * width;

		int top_offset = (n * channels + c) * pooled_height * pooled_width;
		const Dtype* offset_top_diff = top_diff + top_offset;
		const Dtype top_diff_this_bin = offset_top_diff[ph * pooled_width + pw];

		// We use roi_bin_grid to sample the grid and mimic integral
		int roi_bin_grid_h = (sampling_ratio > 0)
			? sampling_ratio
			: ceil(roi_height / pooled_height); // e.g., = 2
		int roi_bin_grid_w =
			(sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

		// We do average (integral) pooling inside a bin
		const Dtype count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

		for (int iy = 0; iy < roi_bin_grid_h; iy++) // e.g., iy = 0, 1
		{
			const Dtype y = roi_start_h + ph * bin_size_h +
				static_cast<Dtype>(iy + .5f) * bin_size_h /
				static_cast<Dtype>(roi_bin_grid_h); // e.g., 0.5, 1.5
			for (int ix = 0; ix < roi_bin_grid_w; ix++) {
				const Dtype x = roi_start_w + pw * bin_size_w +
					static_cast<Dtype>(ix + .5f) * bin_size_w /
					static_cast<Dtype>(roi_bin_grid_w);

				Dtype w1, w2, w3, w4;
				int x_low, x_high, y_low, y_high;

				bilinear_interpolate_gradient(
					height,
					width,
					y,
					x,
					w1,
					w2,
					w3,
					w4,
					x_low,
					x_high,
					y_low,
					y_high,
					index);

				Dtype g1 = top_diff_this_bin * w1 / count;
				Dtype g2 = top_diff_this_bin * w2 / count;
				Dtype g3 = top_diff_this_bin * w3 / count;
				Dtype g4 = top_diff_this_bin * w4 / count;

				if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0) {
					caffe_gpu_atomic_add(
						static_cast<Dtype>(g1), offset_bottom_diff + y_low * width + x_low);
					caffe_gpu_atomic_add(
						static_cast<Dtype>(g2), offset_bottom_diff + y_low * width + x_high);
					caffe_gpu_atomic_add(
						static_cast<Dtype>(g3), offset_bottom_diff + y_high * width + x_low);
					caffe_gpu_atomic_add(
						static_cast<Dtype>(g4), offset_bottom_diff + y_high * width + x_high);
				} // if
			} // ix
		} // iy
	} // CUDA_1D_KERNEL_LOOP
} // RoIAlignBackward




template <typename Dtype>
void ROIAlignLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* bottom_rois = bottom[1]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  const int top_count = top[0]->count();
  const int roi_num = bottom[1]->num();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  

  RoIAlignBackwardFeature<Dtype> << <CAFFE_GET_BLOCKS(count),
	  CAFFE_CUDA_NUM_THREADS >> >(
	  top_count,
	  top_diff,
	  roi_num,
	  spatial_scale_,
	  channels_,
	  height_,
	  width_,
	  pooled_height_,
	  pooled_width_,
	  sampling_ratio_,
	  bottom_diff,
	  bottom_rois);


  // NOLINT_NEXT_LINE(whitespace/operators)
  //ROIPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
  //    count, top_diff, argmax_data, top[0]->num(), spatial_scale_, channels_,
  //    height_, width_, pooled_height_, pooled_width_, bottom_diff, bottom_rois);
  CUDA_POST_KERNEL_CHECK;
}
//
INSTANTIATE_LAYER_GPU_FUNCS(ROIAlignLayer);

}  // namespace caffe
